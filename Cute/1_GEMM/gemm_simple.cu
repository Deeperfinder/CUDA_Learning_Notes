#include "hip/hip_runtime.h"
#include <cute/tensor.hpp>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <iostream>
#include <cstdlib> // For std::atoi
#include <vector>

template<typename T>
void gen_rand_data(T *data, int n);

template<typename T, int kTileM, int kTileN, int kTileK, typename TiledMMA>
__global__ void gemm_simple(T *Cptr, T *Aptr, T *Bptr, int m, int n, int k){
    using namespace cute;
    Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
    Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
    Tensor C = make_tensor(make_gmem_ptr(Cptr), make_shape(m, n), make_stride(n, Int<1>{}));

    int ix = blockIdx.x;
    int iy = blockIdx.y;
    // Tile 级别分解
    // 坐标为第四象限,这里gB的make_coord(ix, _)即为取出B tile中的一行，其shape 为(kTileN, k)
    // 或者可以记为(kTileN, kTileK, num_tile_k)
    // 注意这里的"_" 符号为切分第ix 行，然后以KtileM和KtileK为小矩阵shape，重复按照这个shape切分多个，获得num_tile_k个tile
    /*
    exmaple:
        tensor A layout:
            ptr[16b](0x56237c284640) o (4,8):(8,1):
                0.00    1.00    2.00    3.00    4.00    5.00    6.00    7.00
                8.00    9.00   10.00   11.00   12.00   13.00   14.00   15.00
                16.00   17.00   18.00   19.00   20.00   21.00   22.00   23.00
                24.00   25.00   26.00   27.00   28.00   29.00   30.00   31.00

        local_tile(A, make_tile(2,2),make_coord(0,0)):
            ptr[16b](0x56237c284640) o (2,2):(8,1):
                0.00    1.00
                8.00    9.00
        
        local_tile(A, make_tile(2,2),make_coord(0,_));
            ptr[16b](0x56237c284640) o (2,2,4):(8,1,2):
                0.00    1.00
                8.00    9.00
            ----------
                2.00    3.00
                10.00   11.00
            ----------
                4.00    5.00
                12.00   13.00
            ----------
                6.00    7.00
                14.00   15.00
    */
    Tensor gA = local_tile(A, make_tile(Int<kTileM>{}, Int<kTileK>{}), make_coord(iy, _)); //(BM,BK,Num_tilek)
    Tensor gB = local_tile(B, make_tile(Int<kTileN>{}, Int<kTileK>{}), make_coord(ix, _)); //(BN,BK,Num_tilek)
    Tensor gC = local_tile(C, make_tile(Int<kTileM>{}, Int<kTileN>{}), make_coord(iy, ix)); // (BM,BN)

    // thread级别分解
    // MMA: TiledMMA一次能做的矩阵运算所需要的数据
    // MMA_M, MMA_K 表示(kTileM, kTileK)按照TiledMMA能力划分的时候，M方向和K方向需要重复多少次TiledMMA才能完成该矩阵乘法                                                                                  

    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);
    auto tAgA = thr_mma.partition_A(gA);    //(MMA, MMA_M, MMA_K)
    auto tBgB = thr_mma.partition_B(gB);    //(MMA, MMA_N, MMA_K)
    auto tCgC = thr_mma.partition_C(gC);    //(MMA, MMA_M, MMA_N)

    auto tArA = thr_mma.partition_fragment_A(gA(_, _, 0));
    auto tBrB = thr_mma.partition_fragment_B(gB(_, _, 0));
    auto tCrC = thr_mma.partition_fragment_C(gC(_, _));
    
    clear(tCrC);
    // 即有多少个k需要在一个block中进行计算。
    int num_tile_k = size<2>(gA); 
#pragma unroll 1
    for(int itile=0; itile < num_tile_k; ++itile){
        // Global mem -> register mem
        cute::copy(tAgA(_, _, _, itile), tArA);
        cute::copy(tBgB(_, _, _, itile), tBrB);

        cute::gemm(tiled_mma, tCrC, tArA, tBrB, tCrC);
    }
    cute::copy(tCrC, tCgC);
}
//gemm_simple<T, kTileM, kTileN, kTileK, MMA><<<grid, block>>>(Cptr, Aptr, Bptr, m, n, k);
template<typename T, int KTileM, int KTileN, int KTileK, typename TiledMMA>    
void launch_hgemm_simple_cute_wrapper(T *Cptr, T *Aptr, T *Bptr, int m, int n, int k){
    dim3 block(size(TiledMMA{}));
    dim3 grid(n / KTileN, m / KTileM);
    gemm_simple<T, KTileM, KTileN, KTileK, TiledMMA><<<grid, block>>>(Cptr, Aptr, Bptr, m, n, k);
}

template<typename T>
float perf_gemm_swizzle(void (*gpu_hgemm)(T*, T*, T*, int, int, int),
                  int m, int n, int k, int inner_repeat, int warm_up = 2){
    // 初始化data
    // 传入指针以便修改Aptr的内存地址

    T *Cptr;
    T *Aptr;
    T *Bptr;

    T *Cptr_host;
    T *Aptr_host;
    T *Bptr_host;
    // 这里也可以使用hipMalloc直接赋值
    // hipMalloc(&Cptr, size_a);
    hipMalloc(&Aptr, sizeof(T) * m * k);
    hipMalloc(&Bptr, sizeof(T) * n * k);
    hipMalloc(&Cptr, sizeof(T) * m * n);

    Aptr_host = (T*)malloc(sizeof(T) * m * k);
    Bptr_host = (T*)malloc(sizeof(T) * n * k);
    Cptr_host = (T*)malloc(sizeof(T) * m * n);

    gen_rand_data(Aptr_host, m*k);
    gen_rand_data(Bptr_host, n*k);

    hipMemcpy(Aptr, Aptr_host, sizeof(T) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(Bptr, Bptr_host, sizeof(T) * n * k, hipMemcpyHostToDevice); 

    // warmup
    for(int i=0; i < warm_up; i++){
        gpu_hgemm(Cptr, Aptr, Bptr, m, n, k);
    }
    hipDeviceSynchronize();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for(int i =0; i < inner_repeat; i++){
        gpu_hgemm(Cptr, Aptr, Bptr, m, n, k);
    }
    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipEventSynchronize(stop);
    
    float msec, sec;
    hipEventElapsedTime(&msec, start, stop);
    sec = msec / 1000.0 / inner_repeat;

    hipFree(Aptr);
    hipFree(Bptr);
    hipFree(Cptr);
    free(Aptr_host);
    free(Bptr_host);
    free(Cptr_host);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return sec;
}

int main(int argc, char *argv[]){
    // 基础设置
    srand(10086);
    using namespace cute;
    using T = half_t;

    // 设置默认的矩阵大小
    int m = 5120;
    int n = 5120;
    int k = 5120;

    // 检查并解析命令行参数
    if (argc > 1) {
        m = std::atoi(argv[1]);
    }
    if (argc > 2) {
        n = std::atoi(argv[2]);
    }
    if (argc > 3) {
        k = std::atoi(argv[3]);
    }
    
    // 打印最终的矩阵大小以供确认
    std::cout << "Using matrix dimensions: M=" << m << ", N=" << n << ", K=" << k << std::endl;

    // 定义重复次数
    int outer_repeat = 10, inner_repeat = 10;
    double max_sec = 0.0;
    double min_sec = __DBL_MAX__;
    double total_sec = 0.0;

    // 定义了一个Tensor core矩阵乘加操作类型，
    // 16 * 8 * 16（MNK）：-M=16, -K=16, -N=8
    // F16F16F16F16: 表示输入A(F16), 输入B（F16), 累加器C（F16）, 输出D(F16) 均为半精度浮点数 
    using mma_op = SM80_16x8x16_F16F16F16F16_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    // 定义如何将一个更大的MMA操作分解成由Tensor Core或者其他硬件单元执行的更小的原子MMA操作
    // A 矩阵分块(M/2, K/2)
    // B 矩阵分块(M, K/2)
    using MMA = decltype(make_tiled_mma(mma_atom{},
                        make_layout(Shape<_2, _2, _1>{}),
                        make_layout(Shape<_1, _2, _1>{})));
    constexpr int kTileM = 128;
    constexpr int kTileN = 128;
    constexpr int kTileK = 32;
    // printf("MMA: %i\n", static_cast<int>(size(MMA{})));
    // (128, 1, 1)

    for(int i=0; i < outer_repeat; ++i){
        double this_sec = perf_gemm_swizzle<T>(launch_hgemm_simple_cute_wrapper<T,kTileM,kTileN,kTileK, MMA>, m,n,k, inner_repeat);
        max_sec = max(max_sec, this_sec);
        min_sec = min(min_sec, this_sec);
        total_sec += this_sec;
    }
    double avg_sec = total_sec / outer_repeat;
    double avg_Tflops = ((double)m) * n * k * 2 * 1e-12 / avg_sec;
    double achieveUsage = avg_Tflops / 125.0;
    printf("[log] M N K = %6d %6d %6d , \n", m, n, k);
    printf("[log] min_time = %12.8lf s , avg_time = %12.8lf, max_time =  %12.8lf s, \n", min_sec, avg_sec, max_sec);
    printf("[log Cute] HardWare Peak BF16 Performance = %d Tflops,  AVG Performance = %3.4lf Tflops, achieve usage = %f \n", 125, avg_Tflops, achieveUsage);


    hipDeviceSynchronize();
    auto err = hipGetLastError();
    printf("err = %d, str = %s \n", err, hipGetErrorString(err));

    //  cublas
    T *Cptr_cublas;
    hipMalloc(&Cptr_cublas, sizeof(T) *m *n);
    T *Aptr;
    T *Bptr;
    hipMalloc(&Aptr, sizeof(T) * m * k);
    hipMalloc(&Bptr, sizeof(T) * n * k);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    half alpha = half(1.f);
    half beta = half(0.f);
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);
    // warm up
    hipblasStatus_t ret = hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                        n, m, k,
                                        &alpha,
                                        (half *)Bptr, k,
                                        (half *)Aptr, k,
                                        &beta,
                                        (half *)Cptr_cublas, n);
    for (int i = 0; i < 500; ++i) {
         hipblasStatus_t ret = hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                                            n, m, k,
                                            &alpha,
                                            (half *)Bptr, k,
                                            (half *)Aptr, k,
                                            &beta,
                                            (half *)Cptr_cublas, n);
        // if (ret != HIPBLAS_STATUS_SUCCESS) {
        // printf("blas err = %d, str = %s\n", ret, cublasGetStatusString(ret));
        // }
    }
    hipEventRecord(end);
    hipDeviceSynchronize();
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / 500;
    double cublas_Tflops = (2.0 * m * n * k) * 1e-12 / sec;
    double cublas_AchieveUsage = (double)cublas_Tflops / 125.0;
    printf("[log cublas] HardWare Peak BF16 Performance = %d Tflops,  AVG Performance = %3.4lf Tflops, achieve usage = %f \n", 125, cublas_Tflops, cublas_AchieveUsage);
    
    hipFree(Aptr);
    hipFree(Bptr);
    hipFree(Cptr_cublas);
    // hipDeviceSynchronize();
    // err = hipGetLastError();
    // printf("err = %d, str = %s\n", err, hipGetErrorString(err));
    
    // T *Cptr_cublas_host;
    // Cptr_cublas_host = (T*)malloc(sizeof(T) * m * n);

    // //compare 
    // hipMemcpy(Cptr_host, Cptr, sizeof(T) *m * n, hipMemcpyDeviceToHost);
    // hipMemcpy(Cptr_cublas_host, Cptr_cublas, sizeof(T)*m*n, hipMemcpyDeviceToHost);

    
    // float threshold = 0.1;
    // for (int i = 0; i < m * n; ++i) {
    //     float v1 = Cptr_host[i];
    //     float v2 = Cptr_cublas_host[i];
    //     if (fabs(v2 - v1) > threshold) {
    //     printf("v1 = %f, v2 = %f\n", v1, v2);
    //     }
    // }
}
template<typename T>
void gen_rand_data(T *data, int n){
    for(int i=0; i<n; i++){
        float v = (rand() % 200 -100) * 0.01;
        data[i] = v;
    }
}